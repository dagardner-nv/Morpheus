#include <hip/hip_runtime.h>
#include "matx.h"
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>

#include <cassert>
#include <cstdint>
#include <iostream>
#include <memory>
#include <ostream>
#include <vector>

void row_major_test()
{
    // row major 2d [2, 4]
    std::vector<double> input{
        0.1, 0.7, 0.7, 0.7,
        1.0, 0.9, 0.5, 0.9,
        1.0, 0.6, 0.2, 0.6,
        1.0, 0.2, 0.3, 0.2,
    };

    std::vector<double> expected_output{
        1.0, 0.9, 0.7, 0.9,
        1.0, 0.6, 0.3, 0.6
    };

    matx::index_t num_cols      = 4;
    matx::index_t num_rows      = 4;
    matx::index_t expected_rows = expected_output.size() / num_cols;

    assert((num_cols * num_rows) == input.size());
    assert(expected_rows == 2);

    matx::index_t buff_size = input.size() * sizeof(double);
    auto input_buffer     = std::make_shared<rmm::device_buffer>(buff_size, rmm::cuda_stream_per_thread);
    hipMemcpy(input_buffer->data(), input.data(), input_buffer->size(), hipMemcpyHostToDevice);

    auto output_buffer = std::make_shared<rmm::device_buffer>(expected_rows * num_cols * sizeof(double), input_buffer->stream(), input_buffer->memory_resource());

    // collapse rows 0 & 1 together
    {
        auto input_ptr = static_cast<double*>(input_buffer->data());
        auto output_ptr = static_cast<double*>(output_buffer->data());

        matx::DefaultDescriptor<2> input_desc{{2, num_cols},
                                              {num_cols, 1}};

        matx::DefaultDescriptor<1> output_desc{{num_cols}, {1}};

        auto input_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(input_ptr, std::move(input_desc));
        auto output_tensor = matx::make_tensor<double, matx::DefaultDescriptor<1>>(output_ptr, std::move(output_desc));

        matx::rmax(output_tensor, input_tensor.Permute({1, 0}), output_buffer->stream().value());
    }

    // collapse rows 2 & 3 together
    {
        auto input_ptr = static_cast<double*>(input_buffer->data()) + (2 * num_cols);
        auto output_ptr = static_cast<double*>(output_buffer->data()) + (1 * num_cols);

        matx::DefaultDescriptor<2> input_desc{{2, num_cols},
                                              {num_cols, 1}};

        matx::DefaultDescriptor<1> output_desc{{num_cols}, {1}};

        auto input_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(input_ptr, std::move(input_desc));
        auto output_tensor = matx::make_tensor<double, matx::DefaultDescriptor<1>>(output_ptr, std::move(output_desc));

        matx::rmax(output_tensor, input_tensor.Permute({1, 0}), output_buffer->stream().value());
    }

    std::vector<double> host_output(expected_output.size());
    hipMemcpy(host_output.data(), output_buffer->data(), output_buffer->size(), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        assert(host_output[i] == expected_output[i]);
    }
}

void col_major_test()
{
    // col major 2d [4, 4]
    std::vector<double> input{
        0.1, 1.0, 1.0, 1.0,
        0.7, 0.9, 0.6, 0.2,
        0.7, 0.5, 0.2, 0.3,
        0.7, 0.9, 0.6, 0.2
    };

    std::vector<double> expected_output{
        1.0, 1.0,
        0.9, 0.6,
        0.7, 0.3,
        0.9, 0.6
    };

    matx::index_t num_cols      = 4;
    matx::index_t num_rows      = 4;
    matx::index_t expected_rows = expected_output.size() / num_cols;

    assert((num_cols * num_rows) == input.size());
    assert(expected_rows == 2);

    matx::index_t buff_size = input.size() * sizeof(double);
    auto input_buffer     = std::make_shared<rmm::device_buffer>(buff_size, rmm::cuda_stream_per_thread);
    hipMemcpy(input_buffer->data(), input.data(), input_buffer->size(), hipMemcpyHostToDevice);

    auto tmp_buffer = std::make_shared<rmm::device_buffer>(expected_rows * num_cols * sizeof(double), input_buffer->stream(), input_buffer->memory_resource());

    // collapse rows 0 & 1 together
    {
        auto input_ptr = static_cast<double*>(input_buffer->data());
        auto output_ptr = static_cast<double*>(tmp_buffer->data());

        matx::DefaultDescriptor<2> input_desc{{2, num_cols},
                                              {1, num_rows}};

        matx::DefaultDescriptor<1> output_desc{{num_cols}, {1}};

        auto input_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(input_ptr, std::move(input_desc));
        auto output_tensor = matx::make_tensor<double, matx::DefaultDescriptor<1>>(output_ptr, std::move(output_desc));

        matx::rmax(output_tensor, input_tensor.Permute({1, 0}), tmp_buffer->stream().value());
    }

    std::vector<double> host_output(expected_output.size());
    hipMemcpy(host_output.data(), tmp_buffer->data(), tmp_buffer->size(), hipMemcpyDeviceToHost);

    /*
    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        std::cerr << "i= " << i << " v= " << host_output[i] << std::endl << std::flush;
    }
    std::cerr << "\n-----------\n";
    */

    // collapse rows 2 & 3 together
    {
        auto input_ptr = static_cast<double*>(input_buffer->data()) + (2);
        auto output_ptr = static_cast<double*>(tmp_buffer->data()) + (1 * num_cols);

        matx::DefaultDescriptor<2> input_desc{{2, num_cols},
                                              {1, num_rows}};

        matx::DefaultDescriptor<1> output_desc{{num_cols}, {1}};

        auto input_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(input_ptr, std::move(input_desc));
        auto output_tensor = matx::make_tensor<double, matx::DefaultDescriptor<1>>(output_ptr, std::move(output_desc));

        matx::rmax(output_tensor, input_tensor.Permute({1, 0}), tmp_buffer->stream().value());
    }

    hipMemcpy(host_output.data(), tmp_buffer->data(), tmp_buffer->size(), hipMemcpyDeviceToHost);

    /*
    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        std::cerr << "i= " << i << " v= " << host_output[i] << std::endl << std::flush;
    }
    std::cerr << "\n-----------\n";
    */

    auto output_buffer = std::make_shared<rmm::device_buffer>(expected_rows * num_cols * sizeof(double), input_buffer->stream(), input_buffer->memory_resource());
    // copy the row-major tmp_buffer to the output_buffer in column major
    {
        auto tmp_ptr = static_cast<double*>(tmp_buffer->data());
        auto output_ptr = static_cast<double*>(output_buffer->data());

        matx::DefaultDescriptor<2> tmp_desc{{expected_rows, num_cols},
                                              {num_cols, 1}};

        matx::DefaultDescriptor<2> output_desc{{expected_rows, num_cols}, {1, expected_rows}};

        auto tmp_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(tmp_ptr, std::move(tmp_desc));
        auto output_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(output_ptr, std::move(output_desc));

        (output_tensor = tmp_tensor).run(tmp_buffer->stream().value());
    }

    hipMemcpy(host_output.data(), output_buffer->data(), output_buffer->size(), hipMemcpyDeviceToHost);

    /*
    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        std::cerr << "i= " << i << " v= " << host_output[i] << std::endl << std::flush;
    }
    */

    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        assert(host_output[i] == expected_output[i]);
    }
}


void col_major_slice_test()
{
    // col major 2d [4, 4]
    std::vector<double> input{
        0.1, 1.0, 1.0, 1.0,
        0.7, 0.9, 0.6, 0.2,
        0.7, 0.5, 0.2, 0.3,
        0.7, 0.9, 0.6, 0.2
    };

    std::vector<double> expected_output{
        1.0, 1.0,
        0.9, 0.6,
        0.7, 0.3,
        0.9, 0.6
    };

    matx::index_t num_cols      = 4;
    matx::index_t num_rows      = 4;
    matx::index_t expected_rows = expected_output.size() / num_cols;

    assert((num_cols * num_rows) == input.size());
    assert(expected_rows == 2);

    matx::index_t buff_size = input.size() * sizeof(double);
    auto input_buffer     = std::make_shared<rmm::device_buffer>(buff_size, rmm::cuda_stream_per_thread);
    hipMemcpy(input_buffer->data(), input.data(), input_buffer->size(), hipMemcpyHostToDevice);

    auto output_buffer = std::make_shared<rmm::device_buffer>(expected_rows * num_cols * sizeof(double), input_buffer->stream(), input_buffer->memory_resource());

    auto input_ptr = static_cast<double*>(input_buffer->data());
    auto output_ptr = static_cast<double*>(output_buffer->data());

    matx::DefaultDescriptor<2> input_desc{{num_rows, num_cols}, {1, num_rows}};
    matx::DefaultDescriptor<2> output_desc{{expected_rows, num_cols}, {1, expected_rows}};

    auto input_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(input_ptr, std::move(input_desc));
    auto output_tensor = matx::make_tensor<double, matx::DefaultDescriptor<2>>(output_ptr, std::move(output_desc));

    input_tensor.Print();
    std::cerr << "-----------\n";
    output_tensor.Print();

    // collapse rows 0 & 1 together
    {
        auto input_slice = input_tensor.Slice({0, 0}, {2, matx::matxEnd});
        std::cerr << "-----------\nInput Slice\n";
        input_slice.Print();
        std::cerr << "-----------\n";
        auto output_slice = output_tensor.Slice<1>({0, 0}, {matx::matxDropDim, matx::matxEnd}, {1, expected_rows});
        std::cerr << "-----------\nOutput Slice\n";
        output_slice.Print();
        std::cerr << "-----------\n";

        matx::rmax(output_slice, input_slice.Permute({1, 0}), output_buffer->stream().value());
        std::cerr << "-----------\nOutput Slice\n";
        output_slice.Print();
        std::cerr << "-----------\n";
    }

    std::cerr << "-----------\n";
    output_tensor.Print();

    std::vector<double> host_output(expected_output.size());
    hipMemcpy(host_output.data(), output_buffer->data(), output_buffer->size(), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        std::cerr << "i= " << i << " v= " << host_output[i] << std::endl << std::flush;
    }
    std::cerr << "\n-----------\n";

    // collapse rows 2 & 3 together
    {
        auto input_slice = input_tensor.Slice({2, 0}, {4, matx::matxEnd});
        auto output_slice = output_tensor.Slice<1>({1, 0}, {matx::matxDropDim, matx::matxEnd}, {expected_rows});

        matx::rmax(output_slice, input_slice.Permute({1, 0}), output_buffer->stream().value());
    }

    hipMemcpy(host_output.data(), output_buffer->data(), output_buffer->size(), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        std::cerr << "i= " << i << " v= " << host_output[i] << std::endl << std::flush;
    }
    std::cerr << "\n-----------\n";

    for (std::size_t i = 0; i < host_output.size(); ++i)
    {
        assert(host_output[i] == expected_output[i]);
    }
}


int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
    row_major_test();
    std::cout << "Row Major test passed" << std::endl;

    col_major_test();
    std::cout << "Col Major test passed" << std::endl;

    col_major_slice_test();
    std::cout << "Col Major Slice test passed" << std::endl;

    return 0;
}
